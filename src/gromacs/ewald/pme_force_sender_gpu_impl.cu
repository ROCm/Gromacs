#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements PME-PP communication using CUDA
 *
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "pme_force_sender_gpu_impl.h"

#include "config.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpueventsynchronizer.h"
#include "gromacs/utility/gmxmpi.h"

namespace gmx
{

/*! \brief Create PME-PP GPU communication object */
PmeForceSenderGpu::Impl::Impl(GpuEventSynchronizer*  pmeForcesReady,
                              MPI_Comm               comm,
                              const DeviceContext&   deviceContext,
                              gmx::ArrayRef<PpRanks> ppRanks) :
    pmeForcesReady_(pmeForcesReady),
    comm_(comm),
    ppRanks_(ppRanks),
    ppCommStream_(ppRanks.size()),
    ppCommEvent_(ppRanks.size()),
    ppCommEventRecorded_(ppRanks.size()),
    deviceContext_(deviceContext),
    pmeRemoteCpuForcePtr_(ppRanks.size()),
    pmeRemoteGpuForcePtr_(ppRanks.size())
{
    // Create streams and events to manage pushing of force buffers to remote PP ranks
    std::unique_ptr<DeviceStream>         stream;
    std::unique_ptr<GpuEventSynchronizer> event;
    size_t                                i = 0;
    for (i = 0; i < ppRanks_.size(); i++)
    {
        stream = std::make_unique<DeviceStream>(deviceContext_, DeviceStreamPriority::High, false);
        ppCommStream_[i] = std::move(stream);
        event            = std::make_unique<GpuEventSynchronizer>();
        ppCommEvent_[i]  = std::move(event);
    }
}

PmeForceSenderGpu::Impl::~Impl() = default;

/*! \brief Sets location of force to be sent to each PP rank  */
void PmeForceSenderGpu::Impl::setForceSendBuffer(DeviceBuffer<Float3> d_f)
{

    // Need to send address to PP rank only for thread-MPI as PP rank pulls
    // data using cudamemcpy
    if (!GMX_THREAD_MPI)
    {
        return;
    }

#if GMX_MPI

    if (localForcePtr_.empty())
    {
        localForcePtr_.resize(ppRanks_.size());
    }
    int ind_start = 0;
    int ind_end   = 0;
    int i         = 0;
    for (const auto& receiver : ppRanks_)
    {
        ind_start = ind_end;
        ind_end   = ind_start + receiver.numAtoms;

        localForcePtr_[i] = &d_f[ind_start];
        // NOLINTNEXTLINE(bugprone-sizeof-expression)
        MPI_Recv(&pmeRemoteGpuForcePtr_[i], sizeof(float3*), MPI_BYTE, receiver.rankId, 0, comm_, MPI_STATUS_IGNORE);
        // NOLINTNEXTLINE(bugprone-sizeof-expression)
        MPI_Recv(&pmeRemoteCpuForcePtr_[i], sizeof(float3*), MPI_BYTE, receiver.rankId, 0, comm_, MPI_STATUS_IGNORE);
        // Send address of event and associated flag to PP rank, to allow remote enqueueing
        // NOLINTNEXTLINE(bugprone-sizeof-expression)
        MPI_Send(&ppCommEvent_[i], sizeof(GpuEventSynchronizer*), MPI_BYTE, receiver.rankId, 0, comm_);

        std::atomic<bool>* tmpPpCommEventRecordedPtr =
                reinterpret_cast<std::atomic<bool>*>(&(ppCommEventRecorded_[i]));
        tmpPpCommEventRecordedPtr->store(false, std::memory_order_release);
        // NOLINTNEXTLINE(bugprone-sizeof-expression)
        MPI_Send(&tmpPpCommEventRecordedPtr, sizeof(std::atomic<bool>*), MPI_BYTE, receiver.rankId, 0, comm_);
        i++;
    }

#else
    GMX_UNUSED_VALUE(d_f);
#endif
}


/*! \brief Send PME synchronizer directly using CUDA memory copy */
void PmeForceSenderGpu::Impl::sendFToPpCudaDirect(int ppRank, int numAtoms, bool sendForcesDirectToPpGpu)
{

    GMX_ASSERT(GMX_THREAD_MPI, "sendFToPpCudaDirect is expected to be called only for Thread-MPI");


#if GMX_MPI
    float3* pmeRemoteForcePtr =
            sendForcesDirectToPpGpu ? pmeRemoteGpuForcePtr_[ppRank] : pmeRemoteCpuForcePtr_[ppRank];

    pmeForcesReady_->enqueueWaitEvent(*ppCommStream_[ppRank]);

    hipError_t stat = hipMemcpyAsync(pmeRemoteForcePtr,
                                       localForcePtr_[ppRank],
                                       numAtoms * sizeof(rvec),
                                       hipMemcpyDefault,
                                       ppCommStream_[ppRank]->stream());
    CU_RET_ERR(stat, "hipMemcpyAsync on Recv from PME CUDA direct data transfer failed");
    ppCommEvent_[ppRank]->markEvent(*ppCommStream_[ppRank]);
    std::atomic<bool>* tmpPpCommEventRecordedPtr =
            reinterpret_cast<std::atomic<bool>*>(&(ppCommEventRecorded_[ppRank]));
    tmpPpCommEventRecordedPtr->store(true, std::memory_order_release);
#else
    GMX_UNUSED_VALUE(ppRank);
    GMX_UNUSED_VALUE(numAtoms);
#endif
}

/*! \brief Send PME data directly using CUDA-aware MPI */
void PmeForceSenderGpu::Impl::sendFToPpCudaMpi(DeviceBuffer<RVec> sendbuf,
                                               int                offset,
                                               int                numBytes,
                                               int                ppRank,
                                               MPI_Request*       request)
{
    GMX_ASSERT(GMX_LIB_MPI, "sendFToPpCudaMpi is expected to be called only for Lib-MPI");

#if GMX_MPI
    // if using GPU direct comm with CUDA-aware MPI, make sure forces are ready on device
    // before sending it to PP ranks
    pmeForcesReady_->waitForEvent();

    MPI_Isend(sendbuf[offset], numBytes, MPI_BYTE, ppRank, 0, comm_, request);

#else
    GMX_UNUSED_VALUE(sendbuf);
    GMX_UNUSED_VALUE(offset);
    GMX_UNUSED_VALUE(numBytes);
    GMX_UNUSED_VALUE(ppRank);
    GMX_UNUSED_VALUE(request);
#endif
}

PmeForceSenderGpu::PmeForceSenderGpu(GpuEventSynchronizer*  pmeForcesReady,
                                     MPI_Comm               comm,
                                     const DeviceContext&   deviceContext,
                                     gmx::ArrayRef<PpRanks> ppRanks) :
    impl_(new Impl(pmeForcesReady, comm, deviceContext, ppRanks))
{
}

PmeForceSenderGpu::~PmeForceSenderGpu() = default;


void PmeForceSenderGpu::setForceSendBuffer(DeviceBuffer<RVec> d_f)
{
    impl_->setForceSendBuffer(d_f);
}

void PmeForceSenderGpu::sendFToPpCudaMpi(DeviceBuffer<RVec> sendbuf,
                                         int                offset,
                                         int                numBytes,
                                         int                ppRank,
                                         MPI_Request*       request)
{
    impl_->sendFToPpCudaMpi(sendbuf, offset, numBytes, ppRank, request);
}

void PmeForceSenderGpu::sendFToPpCudaDirect(int ppRank, int numAtoms, bool sendForcesDirectToPpGpu)
{
    impl_->sendFToPpCudaDirect(ppRank, numAtoms, sendForcesDirectToPpGpu);
}


} // namespace gmx
